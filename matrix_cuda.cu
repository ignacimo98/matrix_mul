#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 250
#define MATRIX_SIZE 4
 
//GPU kernel 
__global__ void gpu_matrix_mult(int *device_a, int *device_b, int *d_result, int n = 4) {
  __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

  int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
  int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
  int tmp = 0;
  int idx;
 
  for (int sub = 0; sub < gridDim.x; ++sub) 
  {
    idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
    if(idx >= n*n)
    {
      // n may not divisible by BLOCK_SIZE
      tile_a[threadIdx.y][threadIdx.x] = 0;
    }
    else
    {
      tile_a[threadIdx.y][threadIdx.x] = device_a[idx];
    }

    idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
    if(idx >= n*n)
    {
      tile_b[threadIdx.y][threadIdx.x] = 0;
    }  
    else
    {
      tile_b[threadIdx.y][threadIdx.x] = device_b[idx];
    }

    __syncthreads();

    // matrix multiplication
    for (int k = 0; k < BLOCK_SIZE; ++k) 
    {
      tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
    }
    __syncthreads();
  }
  if(row < n && col < n)
  {
    d_result[row * n + col] = tmp;
  }
}

int main(int argc, char const *argv[])
{
  printf("Begin \n");

  int *host_a, *host_b, *host_c;
  int *device_a, *device_b, *device_c;

  //memory allocation	
  hipHostMalloc((void **) &host_a, sizeof(int)*MATRIX_SIZE*MATRIX_SIZE);
  hipHostMalloc((void **) &host_b, sizeof(int)*MATRIX_SIZE*MATRIX_SIZE);
  hipHostMalloc((void **) &host_c, sizeof(int)*MATRIX_SIZE*MATRIX_SIZE);

  unsigned int grid_rows = (MATRIX_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
  unsigned int grid_cols = (MATRIX_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
  
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(grid_cols, grid_rows);

  printf("Initialize matrix A\n");
  for (int i = 0; i < MATRIX_SIZE; ++i) {
    for (int j = 0; j < MATRIX_SIZE; ++j) {
      host_a[i * MATRIX_SIZE + j] = i + j;
      printf("%i\t");
    }
    printf("\n");
  }

  printf("Initialize matrix B\n");
  for (int i = 0; i < MATRIX_SIZE; ++i) {
    for (int j = 0; j < MATRIX_SIZE; ++j) {
      host_b[i * MATRIX_SIZE + j] = i + j;
      printf("%i\t");
    }
    printf("\n");
  }

  printf("Allocating device memory...\n");
   //GPU memory allocation
  hipMalloc((void **) &device_a, sizeof(int)*m*MATRIX_SIZE);
  hipMalloc((void **) &device_b, sizeof(int)*MATRIX_SIZE*k);
  hipMalloc((void **) &device_c, sizeof(int)*m*k);

  printf("Copying to device..\n");
  hipMemcpy(device_a, host_a, sizeof(int)*MATRIX_SIZE*MATRIX_SIZE, hipMemcpyHostToDevice);
  hipMemcpy(device_b, host_b, sizeof(int)*MATRIX_SIZE*MATRIX_SIZE, hipMemcpyHostToDevice);

  // Launch kernel 
  gpu_matrix_mult<<<dimGrid, dimBlock>>>(device_a, device_b, device_c, MATRIX_SIZE); 

  //Wait for kernel call to finish
  hipDeviceSynchronize();

  // Transefr results from device to host 
  hipMemcpy(host_c, device_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);

  printf("Reading matrix C\n");
  for (int i = 0; i < MATRIX_SIZE; ++i) {
    for (int j = 0; j < MATRIX_SIZE; ++j) {
      host_c[i * MATRIX_SIZE + j] = i + j;
      printf("%i\t");
    }
    printf("\n");
  }
  
  // free memory
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);
  hipHostFree(host_a);
  hipHostFree(host_b);
  hipHostFree(host_c);
  hipHostFree(h_cc);
  return 0;
}